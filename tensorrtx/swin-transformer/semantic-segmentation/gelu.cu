#include "hip/hip_runtime.h"
#include "gelu.h"
#include <math.h>
namespace nvinfer1
{
    gelu::gelu()
    {
    }

    gelu::~gelu()
    {
    }
    // create the plugin at runtime from a byte stream
    gelu::gelu(const void* data, size_t length)
    {
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        Tn::read(d, mInputSize);
        assert(d == a + length);
    }

    void gelu::serialize(void* buffer) const
    {
        char* d = static_cast<char*>(buffer), *a = d;
        Tn::write(d, mInputSize);
        assert(d == a + getSerializationSize());
    }

    size_t gelu::getSerializationSize() const
    {
        return sizeof(mInputSize);
    }

    int gelu::initialize()
    {
        return 0;
    }

    Dims gelu::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        assert(nbInputDims == 1);
        Dims outputDims;
        outputDims.nbDims = inputs[0].nbDims;
        for (int i = 0; i < inputs[0].nbDims; i++) {
            outputDims.d[i] = inputs[0].d[i];
        }
        return outputDims;
    }

    // Set plugin namespace
    void gelu::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* gelu::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType gelu::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool gelu::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool gelu::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void gelu::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {

        mInputSize = 1;
        for (int i = 0; i < in[0].dims.nbDims; i++) {
            mInputSize *= in[0].dims.d[i];
        }
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void gelu::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void gelu::detachFromContext() {}

    const char* gelu::getPluginType() const
    {
        return "geluLayer_TRT";
    }

    const char* gelu::getPluginVersion() const
    {
        return "1";
    }

    void gelu::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* gelu::clone() const
    {
        gelu *p = new gelu();
        p->setPluginNamespace(mPluginNamespace);
        p->setInputSize(mInputSize);
        return p;
    }

    __global__ void geluKer(const float *in, float *out, int size) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx >= size)
            return;
        //x * 0.5 * (1.0 + torch.erf(x / math.sqrt(2.0)))
        out[idx] = in[idx] * 0.5 *(1.0 + erf(in[idx]/1.4142135381698608));
    }
    void gelu::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {

        int numElem = batchSize * mInputSize;
        geluKer<<<(numElem + mThreadCount - 1) / mThreadCount, mThreadCount>>>
            (inputs[0], output, numElem);
    }

    int gelu::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection geluCreator::mFC{};
    std::vector<PluginField> geluCreator::mPluginAttributes;

    geluCreator::geluCreator()
    {
        mPluginAttributes.clear();
        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* geluCreator::getPluginName() const
    {
            return "geluLayer_TRT";
    }

    const char* geluCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* geluCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* geluCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        gelu* obj = new gelu();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* geluCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        gelu* obj = new gelu(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }


}

